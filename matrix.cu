#include "hip/hip_runtime.h"
#include "matrix.h"


using namespace std;
/*
LIMITE 6144 double ou 49152 bytes 32 THREAD MAX
                                  64 block max
*/

__global__ void multiply(double * A, double * B, double * C,int local_size)
{
    extern __shared__ double shared[];

    double * intern_A = &shared[0];
    double * intern_B = &shared[local_size];
    
    int col =  blockIdx.x * blockDim.x + threadIdx.x;
    int row =  blockIdx.y * blockDim.y + threadIdx.y;

    intern_A[threadIdx.y*blockDim.x + threadIdx.x] = A[row*DIM + threadIdx.x+blockDim.x*blockIdx.z];
    intern_B[threadIdx.y*blockDim.x + threadIdx.x] = B[(threadIdx.y+blockDim.y*blockIdx.z)*DIM + col];

    double result = 0.0;
    for(int i=0; i<blockDim.x; i++)
    {
        result+= intern_A[threadIdx.y*blockDim.x + i]*intern_B[i*blockDim.x  + threadIdx.x];
    }

    C[blockIdx.z*DIM*DIM + row*DIM + col] = result;

}
__global__ void sum(double * C, int n)
{
    int col =  blockIdx.x * blockDim.x + threadIdx.x;
    int row =  blockIdx.y * blockDim.y + threadIdx.y;
    for(int i=1;i<n;i++) C[row*DIM + col] += C[DIM*DIM*i+row*DIM + col];
}

void matrix_multiplication(double * a, double * b, double * c, int dim)
{
    double * result = (double *) malloc(sizeof(double)*dim*dim);

    int blocksNb  = (int) dim/8;
    int threadsNb = (int) 8;
    int sharedMemory = threadsNb*threadsNb*sizeof(double)*2;

    double * d_a; hipMalloc((void **) &d_a, sizeof(double)*dim*dim);
    double * d_b; hipMalloc((void **) &d_b, sizeof(double)*dim*dim);
    double * d_c; hipMalloc((void **) &d_c, sizeof(double)*dim*dim*blocksNb);

    hipMemcpy(d_a, a, sizeof(double)*dim*dim, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(double)*dim*dim, hipMemcpyHostToDevice);

    dim3 multiply_threadsPerBlock(threadsNb, threadsNb);
    dim3 multiply_blocksPerGrid(blocksNb, blocksNb,blocksNb);
    dim3 sum_threadsPerBlock(threadsNb,threadsNb);
    dim3 sum_blocksPerGrid(blocksNb,blocksNb);

    multiply<<<multiply_blocksPerGrid, multiply_threadsPerBlock, sharedMemory>>>(d_a, d_b, d_c, threadsNb*threadsNb);
    sum<<<sum_blocksPerGrid, sum_threadsPerBlock>>>(d_c, blocksNb);

    hipMemcpy(result, d_c, sizeof(double)*dim*dim, hipMemcpyDeviceToHost);

    for(int i=0;i<dim;i++) for(int j=0;j<dim;j++) c[i*dim + j] = result[i*dim + j];

    free(result);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    
}


