#include "hip/hip_runtime.h"
#include "../../include/cuvector.h"
#include "../../include/CUDA_CONSTANT.h"
#include <iostream>


template<typename T> __global__ void cuda_copy(T* dest, T* src)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    dest[index] = src[index];
};


template<typename T>
CuVector<T>::CuVector(int size)
{
    m_maxsize = sizeChecking(size);
    m_size = 0;
    hipMalloc((void **) &m_data, sizeof(T)*size);
}

template<typename T>
CuVector<T>::CuVector(std::vector <T> data)
{
    m_maxsize = sizeChecking(data.capacity());
    m_size = data.size();
    hipMalloc((void **) &m_data, sizeof(T)*m_maxsize);
    hipMemcpy(m_data, data.data(), sizeof(T)*data.size(), hipMemcpyHostToDevice);
}

template<typename T>
CuVector<T>::CuVector(CuVector<T> const& copy)
{
    m_maxsize = sizeChecking(copy.m_maxsize);
    m_size = copy.m_size;
    hipMalloc((void **) &m_data, sizeof(T)*m_maxsize);
    dim3 blocksPerGrid((m_maxsize/MAX_THREAD_BY_BLOCKS)+1,0,0);
    dim3 threadsPerBlocks(m_maxsize/(blocksPerGrid.x),0,0);
    cuda_copy<T><<<blocksPerGrid,threadsPerBlocks>>>(m_data,copy.m_data);
}

template<typename T>
CuVector<T>::~CuVector()
{
    hipFree(m_data);
}



template<typename T>
int CuVector<T>::sizeChecking(int size)
{
    while( size%(size/MAX_THREAD_BY_BLOCKS +1)!=0) size++;
    return size;
}