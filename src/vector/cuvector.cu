#include "hip/hip_runtime.h"
#include "../../include/cuvector.h"
#include "../../include/CUDA_CONSTANT.h"



template<typename T> __global__ void cuda_copy(T* dest, T* src)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    dest[index] = src[index];
};

template<typename T> __global__ void cuda_shift(T* data, int offset)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    T temp = data[index];
    hipDeviceSynchronize();
    data[index+offset] = temp;
}

template<typename T>
CuVector<T>::CuVector(int size)
{
    m_maxsize = sizeChecking(size);
    m_size = 0;
    hipMalloc((void **) &m_data, sizeof(T)*size);
}

template<typename T>
CuVector<T>::CuVector(std::vector <T> data)
{
    m_maxsize = sizeChecking(data.capacity());
    m_size = data.size();
    hipMalloc((void **) &m_data, sizeof(T)*m_maxsize);
    hipMemcpy(m_data, data.data(), sizeof(T)*data.size(), hipMemcpyHostToDevice);
}

template<typename T>
CuVector<T>::CuVector(CuVector<T> const& copy)
{
    m_maxsize = sizeChecking(copy.m_maxsize);
    m_size = copy.m_size;
    hipMalloc((void **) &m_data, sizeof(T)*m_maxsize);
    dim3 blocksPerGrid((m_maxsize/MAX_THREAD_BY_BLOCKS)+1,0,0);
    dim3 threadsPerBlocks(m_maxsize/(blocksPerGrid.x),0,0);
    cuda_copy<T><<<blocksPerGrid,threadsPerBlocks>>>(m_data,copy.m_data);
}

template<typename T>
CuVector<T>::~CuVector()
{
    hipFree(m_data);
}


template<typename T>
void CuVector<T>::show() const
{
    T * local = (T*) malloc(sizeof(T)*this->m_size);
    hipMemcpy(local, m_data, sizeof(T)*this->m_size, hipMemcpyDeviceToHost);
    for(int i=0;i<this->m_size;i++) std::cout<<local[i]<<" ";
    std::cout<<std::endl;
    free(local);
}

template<typename T>
T CuVector<T>::get(int i)
{
    if(i>=m_size) throw std::runtime_error("Index doesn't exist");
    T result;
    hipMemcpy(&result, m_data+i, sizeof(T), hipMemcpyDeviceToHost);
    return result;
}


template<typename T>
T CuVector<T>::pop_back(void)
{
    if(this->empty()) throw std::runtime_error("CuVector empty !");
    T result;
    hipMemcpy(&result, m_data+m_size-1, sizeof(T), hipMemcpyDeviceToHost);
    return result;
}


template <typename T>
T CuVector<T>::pop_front(void)
{
    if(this->empty()) throw std::runtime_error("CuVector empty !");
    T result;
    hipMemcpy(&result, m_data, sizeof(T), hipMemcpyDeviceToHost);
    for(int i=0;i<getBocksSize(m_maxsize); i++)
    {
        if(i==0) cuda_shift<<<1,getThreadsSize(m_maxsize)-1>>>(m_data, i*getThreadsSize(m_maxsize), -1, -1, this->capacity()+1);
        else cuda_shift<<<1,getThreadsSize(m_maxsize)>>>(m_data, i*getThreadsSize(m_maxsize), -1, -1, this->capacity()+1);
        hipDeviceSynchronize();
    }
    m_size--;
    return result;
}

template<typename T>
void CuVector<T>::push_back(const T element)
{
    if(this->full()) throw std::runtime_error("CuVector full !");
    hipMemcpy(m_data + m_size, &element, sizeof(T), hipMemcpyHostToDevice);
    m_size++;

}

template<typename T>
void CuVector<T>::push_front(const T element)
{
    if(this->full()) throw std::runtime_error("CuVector full !");
    //TODO optimisé le nb de boucle
    for(int i=0;i<getBocksSize(m_maxsize); i++)
    {
        if(i==getBocksSize(m_maxsize)-1) cuda_shift<<<1,getThreadsSize(m_maxsize)-1>>>(m_data, i*getThreadsSize(m_maxsize), 1, -1, this->capacity()+1);
        else cuda_shift<<<1,getThreadsSize(m_maxsize)>>>(m_data, i*getThreadsSize(m_maxsize), 1, -1, this->capacity()+1);
        hipDeviceSynchronize();
    }

    hipMemcpy(m_data, &element, sizeof(T), hipMemcpyHostToDevice);

    m_size++;
}

template<typename T>
int CuVector<T>::sizeChecking(int size)
{
    while( size%(size/MAX_THREAD_BY_BLOCKS +1)!=0) size++;
    return size;
}


template <typename T>
int CuVector<T>::getBocksSize(int size)
{
    return ((int) (std::ceil(  ((float) size) / ((float) MAX_THREAD_BY_BLOCKS) )));
}

template <typename T>
int CuVector<T>::getThreadsSize(int size)
{
    // TODO may replace by (m_maxsize/MAX_THREAD_BY_BLOCKS)
    return size/((int) (std::ceil( ((float) size) / ((float) MAX_THREAD_BY_BLOCKS) )));
}


template <typename T>
T CuVector<T>::operator[](int i)
{
    return this->get(i);
}