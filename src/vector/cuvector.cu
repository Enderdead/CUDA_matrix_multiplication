#include "hip/hip_runtime.h"
#include "../../include/cuvector.h"
#include "../../include/CUDA_CONSTANT.h"



template<typename T> __global__ void cuda_copy(T* dest, T* src)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    dest[index] = src[index];
};

template<typename T> __global__ void cuda_shift(T* data, int offset)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    T temp = data[index];
    hipDeviceSynchronize();
    data[index+offset] = temp;
}

template<typename T>
CuVector<T>::CuVector(int size)
{
    m_maxsize = sizeChecking(size);
    m_size = 0;
    hipMalloc((void **) &m_data, sizeof(T)*size);
}

template<typename T>
CuVector<T>::CuVector(std::vector <T> data)
{
    m_maxsize = sizeChecking(data.capacity());
    m_size = data.size();
    hipMalloc((void **) &m_data, sizeof(T)*m_maxsize);
    hipMemcpy(m_data, data.data(), sizeof(T)*data.size(), hipMemcpyHostToDevice);
}

template<typename T>
CuVector<T>::CuVector(CuVector<T> const& copy)
{
    m_maxsize = sizeChecking(copy.m_maxsize);
    m_size = copy.m_size;
    hipMalloc((void **) &m_data, sizeof(T)*m_maxsize);
    dim3 blocksPerGrid((m_maxsize/MAX_THREAD_BY_BLOCKS)+1,0,0);
    dim3 threadsPerBlocks(m_maxsize/(blocksPerGrid.x),0,0);
    cuda_copy<T><<<blocksPerGrid,threadsPerBlocks>>>(m_data,copy.m_data);
}

template<typename T>
CuVector<T>::~CuVector()
{
    hipFree(m_data);
}


template<typename T>
T CuVector<T>::get(int i)
{
    if(i>=m_size) throw std::runtime_errror("Index doesn't exist");
    T result;
    hipMemcpy(&result, m_data+i, sizeof(T), hipMemcpyDeviceToHost);
    return result;
}


template<typename T>
T CuVector<T>::pop_back(void)
{
    if(this->empty()) throw std::runtime_error("CuVector empty !");
    T result;
    hipMemcpy(&result, m_data+m_size-1, sizeof(T), hipMemcpyDeviceToHost);
    return result;
}


template<typename T>
void CuVector<T>::push_back(const T element)
{
    if(this->full()) throw std::runtime_error("CuVector full !");
    hipMemcpy(m_data + m_size, &element, sizeof(T), hipMemcpyHostToDevice);
    m_size++;

}


template<typename T>
int CuVector<T>::sizeChecking(int size)
{
    while( size%(size/MAX_THREAD_BY_BLOCKS +1)!=0) size++;
    return size;
}

template <typename T>
T CuVector<T>::operator[](int i)
{
    return this->get(i);
}